#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 *
 * Edited by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "nvdsinfer_custom_impl.h"

extern "C" bool
NvDsInferParseYoloCuda(std::vector<NvDsInferLayerInfo> const& outputLayersInfo, NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams, std::vector<NvDsInferParseObjectInfo>& objectList);

__global__ void decodeTensorYoloCuda(NvDsInferParseObjectInfo *binfo, const float* output, const uint outputSize,
    const uint netW, const uint netH, const float* preclusterThreshold)
{
  int x_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (x_id >= outputSize) {
    return;
  }

  float maxProb = output[x_id * 6 + 4];
  int maxIndex = (int) output[x_id * 6 + 5];

  if (maxProb < preclusterThreshold[maxIndex]) {
    binfo[x_id].detectionConfidence = 0.0;
    return;
  }

  float bx1 = output[x_id * 6 + 0];
  float by1 = output[x_id * 6 + 1];
  float bx2 = output[x_id * 6 + 2];
  float by2 = output[x_id * 6 + 3];

  bx1 = fminf(float(netW), fmaxf(float(0.0), bx1));
  by1 = fminf(float(netH), fmaxf(float(0.0), by1));
  bx2 = fminf(float(netW), fmaxf(float(0.0), bx2));
  by2 = fminf(float(netH), fmaxf(float(0.0), by2));

  binfo[x_id].left = bx1;
  binfo[x_id].top = by1;
  binfo[x_id].width = fminf(float(netW), fmaxf(float(0.0), bx2 - bx1));
  binfo[x_id].height = fminf(float(netH), fmaxf(float(0.0), by2 - by1));
  binfo[x_id].detectionConfidence = maxProb;
  binfo[x_id].classId = maxIndex;
}

static bool NvDsInferParseCustomYoloCuda(std::vector<NvDsInferLayerInfo> const& outputLayersInfo,
    NvDsInferNetworkInfo const& networkInfo, NvDsInferParseDetectionParams const& detectionParams,
    std::vector<NvDsInferParseObjectInfo>& objectList)
{
  if (outputLayersInfo.empty()) {
    std::cerr << "ERROR: Could not find output layer in bbox parsing" << std::endl;
    return false;
  }

  const NvDsInferLayerInfo& output = outputLayersInfo[0];
  const uint outputSize = output.inferDims.d[0];

  thrust::device_vector<float> perClassPreclusterThreshold = detectionParams.perClassPreclusterThreshold;

  thrust::device_vector<NvDsInferParseObjectInfo> objects(outputSize);

  int threads_per_block = 1024;
  int number_of_blocks = ((outputSize) / threads_per_block) + 1;

  decodeTensorYoloCuda<<<number_of_blocks, threads_per_block>>>(
      thrust::raw_pointer_cast(objects.data()), (float*) (output.buffer), outputSize, networkInfo.width,
          networkInfo.height, thrust::raw_pointer_cast(perClassPreclusterThreshold.data()));

  objectList.resize(outputSize);
  thrust::copy(objects.begin(), objects.end(), objectList.begin());

  return true;
}

extern "C" bool
NvDsInferParseYoloCuda(std::vector<NvDsInferLayerInfo> const& outputLayersInfo, NvDsInferNetworkInfo const& networkInfo,
    NvDsInferParseDetectionParams const& detectionParams, std::vector<NvDsInferParseObjectInfo>& objectList)
{
  return NvDsInferParseCustomYoloCuda(outputLayersInfo, networkInfo, detectionParams, objectList);
}

CHECK_CUSTOM_PARSE_FUNC_PROTOTYPE(NvDsInferParseYoloCuda);
